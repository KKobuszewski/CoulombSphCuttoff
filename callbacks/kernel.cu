#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <assert.h>

#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

//#define DEBUG

#define BLOCKSIZE 256

#define NN 16

    /**********/
    /* iDivUp */
    /**********/
    int iDivUp(int a, int b) { return ((a % b) != 0) ? (a / b + 1) : (a / b); }

    /********************/
    /* CUDA ERROR CHECK */
    /********************/
    #define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
    inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
    {
       if (code != hipSuccess)
       {
          fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
          if (abort) exit(code);
       }
    }

    /*********************/
    /* CUFFT ERROR CHECK */
    /*********************/
    // See http://stackoverflow.com/questions/16267149/cufft-error-handling

    #ifdef _CUFFT_H_
    // cuFFT API errors
    static const char *_cudaGetErrorEnum(hipfftResult error)
    {
        switch (error)
        {
            case HIPFFT_SUCCESS:
                return "HIPFFT_SUCCESS";

            case HIPFFT_INVALID_PLAN:
                return "HIPFFT_INVALID_PLAN";

            case HIPFFT_ALLOC_FAILED:
                return "HIPFFT_ALLOC_FAILED";

            case HIPFFT_INVALID_TYPE:
                return "HIPFFT_INVALID_TYPE";

            case HIPFFT_INVALID_VALUE:
                return "HIPFFT_INVALID_VALUE";

            case HIPFFT_INTERNAL_ERROR:
                return "HIPFFT_INTERNAL_ERROR";

            case HIPFFT_EXEC_FAILED:
                return "HIPFFT_EXEC_FAILED";

            case HIPFFT_SETUP_FAILED:
                return "HIPFFT_SETUP_FAILED";

            case HIPFFT_INVALID_SIZE:
                return "HIPFFT_INVALID_SIZE";

            case HIPFFT_UNALIGNED_DATA:
                return "HIPFFT_UNALIGNED_DATA";
        }

        return "<unknown>";
    }
    #endif

    #define cufftSafeCall(err)      __cufftSafeCall(err, __FILE__, __LINE__)
    inline void __cufftSafeCall(hipfftResult err, const char *file, const int line)
    {
        if( HIPFFT_SUCCESS != err) {
    		fprintf(stderr, "CUFFT error in file '%s', line %d\n %s\nerror %d: %s\nterminating!\n",__FILE__, __LINE__,err, \
                                    _cudaGetErrorEnum(err)); \
    		hipDeviceReset(); assert(0); \
    	}
    }

    /****************************************/
    /* FFTSHIFT 1D INPLACE MEMORY MOVEMENTS */
    /****************************************/
    __global__ void fftshift_1D_inplace_memory_movements(float2* d_inout, unsigned int N)
    {
    	unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;

    	if (tid < N/2)
        {
    		printf("%f %f\n", 2.f*d_inout[tid].x, 3.f*d_inout[tid].x);

    		float2 temp = d_inout[tid];
            d_inout[tid] = d_inout[tid + (N / 2)];
            d_inout[tid + (N / 2)] = temp;
        }
    }

    /**********************************************/
    /* FFTSHIFT 1D INPLACE CHESSBOARD - VERSION 1 */
    /**********************************************/
    __device__ float2 fftshift_1D_chessboard_callback_v1(void *d_in, size_t offset, void *callerInfo, void *sharedPtr) {

    	float a = (float)(1-2*((int)offset%2));

    	float2	out = ((float2*)d_in)[offset];
    	out.x = out.x * a;
    	out.y = out.y * a;
    	return out;
    }

    __device__ hipfftCallbackLoadC fftshift_1D_chessboard_callback_v1_Ptr = fftshift_1D_chessboard_callback_v1;

    /********/
    /* MAIN */
    /********/
    int main()
    {
        const int N = 16;

    	// --- Host side input array
    	float2 *h_vect = (float2 *)malloc(N*sizeof(float2));
    	for (int i=0; i<N; i++) {
    		h_vect[i].x = (float)rand() / (float)RAND_MAX;
    		h_vect[i].y = (float)rand() / (float)RAND_MAX;
    	}

    	// --- Host side output arrays
    	float2 *h_out1 = (float2 *)malloc(N*sizeof(float2));
    	float2 *h_out2 = (float2 *)malloc(N*sizeof(float2));

    	// --- Device side input arrays
    	float2 *d_vect1; gpuErrchk(hipMalloc((void**)&d_vect1, N*sizeof(float2)));
    	float2 *d_vect2; gpuErrchk(hipMalloc((void**)&d_vect2, N*sizeof(float2)));
            gpuErrchk(hipMemcpy(d_vect1, h_vect, N*sizeof(float2), hipMemcpyHostToDevice));
            gpuErrchk(hipMemcpy(d_vect2, h_vect, N*sizeof(float2), hipMemcpyHostToDevice));

    	// --- Device side output arrays
    	float2 *d_out1; gpuErrchk(hipMalloc((void**)&d_out1, N*sizeof(float2)));
    	float2 *d_out2; gpuErrchk(hipMalloc((void**)&d_out2, N*sizeof(float2)));

    	float time;
    	hipEvent_t start, stop;
    	hipEventCreate(&start);
    	hipEventCreate(&stop);

    	/*******************************************/
    	/* cuFFT + MEMORY MOVEMENTS BASED FFTSHIFT */
    	/*******************************************/
    	hipfftHandle planinverse; cufftSafeCall(hipfftPlan1d(&planinverse, N, HIPFFT_C2C, 1));
    	hipEventRecord(start, 0);
    	cufftSafeCall(hipfftExecC2C(planinverse, d_vect1, d_vect1, HIPFFT_BACKWARD));
    	fftshift_1D_inplace_memory_movements<<<iDivUp(N/2, BLOCKSIZE), BLOCKSIZE>>>(d_vect1, N);
    #ifdef DEBUG
    	gpuErrchk(hipPeekAtLastError());
    	gpuErrchk(hipDeviceSynchronize());
    #endif
    	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
    	hipEventElapsedTime(&time, start, stop);
    	printf("Memory movements elapsed time:  %3.3f ms \n", time);
    	gpuErrchk(hipMemcpy(h_out1, d_vect1, N*sizeof(float2), hipMemcpyDeviceToHost));

    	/****************************************/
    	/* CHESSBOARD MULTIPLICATION V1 + cuFFT */
    	/****************************************/
    	hipfftCallbackLoadC hfftshift_1D_chessboard_callback_v1_Ptr;

    	gpuErrchk(hipMemcpyFromSymbol(&hfftshift_1D_chessboard_callback_v1_Ptr, HIP_SYMBOL(fftshift_1D_chessboard_callback_v1_Ptr), sizeof(hfftshift_1D_chessboard_callback_v1_Ptr)));
    	hipfftHandle planinverse_v1; cufftSafeCall(hipfftPlan1d(&planinverse_v1, N, HIPFFT_C2C, 1));
    	hipfftResult status = hipfftXtSetCallback(planinverse_v1, (void **)&hfftshift_1D_chessboard_callback_v1_Ptr, HIPFFT_CB_LD_COMPLEX, 0);
    	if (status == CUFFT_LICENSE_ERROR) {
    	    printf("This sample requires a valid license file.\n");
    	    printf("The file was either not found, out of date, or otherwise invalid.\n");
    	     exit(EXIT_FAILURE);
    	} else {
    		cufftSafeCall(status);
    	  }
    	hipEventRecord(start, 0);
    	cufftSafeCall(hipfftExecC2C(planinverse_v1, d_vect2, d_out2, HIPFFT_BACKWARD));
    	hipEventRecord(stop, 0);
    	hipEventSynchronize(stop);
    	hipEventElapsedTime(&time, start, stop);
    	printf("Chessboard v1 elapsed time:  %3.3f ms \n", time);

    	gpuErrchk(hipMemcpy(h_out2, d_out2, N*sizeof(float2), hipMemcpyDeviceToHost));

    	// --- Checking the results
    	for (int i=0; i<N; i++) if ((h_out1[i].x != h_out2[i].x)||(h_out1[i].y != h_out2[i].y)) { printf("Chessboard v1 test failed!\n"); return 0; }

    	printf("Chessboard v1 test passed!\n");

    	return 0;

    }