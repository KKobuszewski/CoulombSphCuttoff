#include "hip/hip_runtime.h"
/* Copyright (c) 1993-2015, NVIDIA CORPORATION. All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 *  * Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 *  * Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *  * Neither the name of NVIDIA CORPORATION nor the names of its
 *    contributors may be used to endorse or promote products derived
 *    from this software without specific prior written permission.
 *
 * THIS SOFTWARE IS PROVIDED BY THE COPYRIGHT HOLDERS ``AS IS'' AND ANY
 * EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR
 * PURPOSE ARE DISCLAIMED.  IN NO EVENT SHALL THE COPYRIGHT OWNER OR
 * CONTRIBUTORS BE LIABLE FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL,
 * EXEMPLARY, OR CONSEQUENTIAL DAMAGES (INCLUDING, BUT NOT LIMITED TO,
 * PROCUREMENT OF SUBSTITUTE GOODS OR SERVICES; LOSS OF USE, DATA, OR
 * PROFITS; OR BUSINESS INTERRUPTION) HOWEVER CAUSED AND ON ANY THEORY
 * OF LIABILITY, WHETHER IN CONTRACT, STRICT LIABILITY, OR TORT
 * (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY OUT OF THE USE
 * OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF SUCH DAMAGE.
 */

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

// CUDA driver & runtime
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hipfft/hipfftXt.h>

#include "common.h"

#ifdef JIT
#include "ptxjit.h"
#endif // JIT

#define TILE_DIM 32
#define BLOCK_ROWS 8

////////////////////////////////////////////////////////////////////////////////
// Callback Implementations
////////////////////////////////////////////////////////////////////////////////
__device__ hipfftReal CB_ConvertInputR(void *dataIn, size_t offset, void *callerInfo, void *sharedPtr) {
    char element = ((char*)dataIn)[offset];
    return (hipfftReal)((float)element/127.0f);
}

__device__ hipfftCallbackLoadR d_loadCallbackPtr = CB_ConvertInputR; 

__device__ void CB_ConvolveAndStoreTransposedC(void *dataOut, size_t offset, hipfftComplex element, void *callerInfo, void *sharedPtr) {
    hipfftComplex *filter = (hipfftComplex*)callerInfo;
    size_t row = offset / COMPLEX_SIGNAL_SIZE;
    size_t col = offset % COMPLEX_SIGNAL_SIZE;

    ((hipfftComplex*)dataOut)[col * BATCH_SIZE + row] = ComplexMul(element, filter[col]);
}

__device__ hipfftCallbackStoreC d_storeCallbackPtr = CB_ConvolveAndStoreTransposedC;

#ifndef JIT
// postprocessing
__global__ void ConvolveAndStoreTransposedC_Optimized(
    const hipfftComplex * __restrict__ dataIn, 
    hipfftComplex * __restrict__ dataOut,
    const hipfftComplex * __restrict__ filter)
{
    __shared__ hipfftComplex tile[TILE_DIM][TILE_DIM+1];

    int x = blockIdx.x * TILE_DIM + threadIdx.x;
    int yBase = blockIdx.y * TILE_DIM + threadIdx.y;
    
    if(x < COMPLEX_SIGNAL_SIZE) {
        for(int j = 0; j < TILE_DIM; j+= BLOCK_ROWS) {
            int y = yBase + j;
            if(y >= BATCH_SIZE) break;
            hipfftComplex value = ComplexMul(dataIn[y * COMPLEX_SIGNAL_SIZE + x], filter[x]);
            tile[threadIdx.y + j][threadIdx.x] = value;
        }
    }

    __syncthreads();

    x = blockIdx.y * TILE_DIM + threadIdx.x;
    yBase = blockIdx.x * TILE_DIM + threadIdx.y;

    if(x < BATCH_SIZE) {
        for(int j = 0; j < TILE_DIM; j += BLOCK_ROWS) {
            int y = yBase + j;
            if(y >= COMPLEX_SIGNAL_SIZE) break;
            dataOut[y * BATCH_SIZE + x] = tile[threadIdx.x][threadIdx.y + j];
        }  
    }
}
#endif

////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int main(int argc, char **argv)
{
    struct hipDeviceProp_t properties;
    int device = argc > 1 ? atoi(argv[1]) : 0;

    checkCudaErrors(hipGetDevice(&device));
    checkCudaErrors(hipGetDeviceProperties(&properties, device));
    if( !(properties.major >= 2) ) {
        printf("This sample requires CUDA architecture SM2.0 or higher\n");
        exit(EXIT_FAILURE);
    }

    // Allocate and initialize memory
    printf("Preparing input: %dx%d\n", BATCH_SIZE, INPUT_SIGNAL_SIZE);
    char *_8bit_signal;
    hipfftComplex *result, *filter;
    hipfftComplex *tmp_result;

    checkCudaErrors(hipMallocManaged(&_8bit_signal, sizeof(char) * INPUT_SIGNAL_SIZE * BATCH_SIZE, hipMemAttachGlobal));
    checkCudaErrors(hipMallocManaged(&result, sizeof(hipfftComplex) * COMPLEX_SIGNAL_SIZE * BATCH_SIZE, hipMemAttachGlobal));
    checkCudaErrors(hipMallocManaged(&tmp_result, sizeof(hipfftComplex) * COMPLEX_SIGNAL_SIZE * BATCH_SIZE, hipMemAttachGlobal));
    checkCudaErrors(hipMallocManaged(&filter, sizeof(hipfftComplex) * COMPLEX_SIGNAL_SIZE, hipMemAttachGlobal));

    initInputs(_8bit_signal, filter);
    
    //compute reference result for later verification
    printf("Computing reference solution\n");
    hipfftComplex *reference = computeReference(_8bit_signal, filter);

    printf("Creating FFT plan\n");
    hipfftHandle fftPlan;
    size_t workSize;
    
    checkCudaErrors(hipfftCreate(&fftPlan));
    int signalSize = INPUT_SIGNAL_SIZE;
    checkCudaErrors(hipfftMakePlanMany(fftPlan, 1, &signalSize, 0,0,0,0,0,0, HIPFFT_R2C, BATCH_SIZE, &workSize));

    /*
     * Retrieve address of callback functions on the device
     */                              
    hipfftCallbackLoadR h_loadCallbackPtr;
    hipfftCallbackStoreC h_storeCallbackPtr;
    checkCudaErrors(hipMemcpyFromSymbol(&h_loadCallbackPtr, 
                                          HIP_SYMBOL(d_loadCallbackPtr), 
                                          sizeof(h_loadCallbackPtr)));
    checkCudaErrors(hipMemcpyFromSymbol(&h_storeCallbackPtr, 
                                          HIP_SYMBOL(d_storeCallbackPtr), 
                                          sizeof(h_storeCallbackPtr)));

    // Now associate the callbacks with the plan.
    hipfftResult status = hipfftXtSetCallback(fftPlan, 
                            (void **)&h_loadCallbackPtr, 
                            HIPFFT_CB_LD_REAL,
                            0);
    if (status == CUFFT_LICENSE_ERROR) {
        printf("This sample requires a valid license file.\n");
        printf("The file was either not found, out of date, or otherwise invalid.\n");
        exit(EXIT_FAILURE);
    } else {
        checkCudaErrors(status);
    }
    
    
    //checkCudaErrors(hipfftXtSetCallback(fftPlan, (void **)&h_storeCallbackPtr, HIPFFT_CB_ST_COMPLEX, (void **)&filter));
#ifdef JIT
    hipModule_t     hModule  = 0;
    hipFunction_t   hKernel  = 0;
    hiprtcLinkState  lState;
    
    ptxJIT(argc, argv, &hModule, &hKernel, &lState);
#endif
    
    
    //create timers
    hipEvent_t start, end;
    hipEventCreate(&start);
    hipEventCreate(&end);
    float elapsedTime;

    printf("Running %d iterations\n", ITERATIONS);
#ifndef JIT
    checkCudaErrors(hipEventRecord(start, 0));
#endif
    /*
     * The actual Computation
     */
    
    dim3 block(TILE_DIM, BLOCK_ROWS);
    dim3 grid((COMPLEX_SIGNAL_SIZE + block.x - 1)/block.x, (BATCH_SIZE + block.y - 1)/block.y);
    
    for(int i = 0; i < ITERATIONS; i++) 
    {
        checkCudaErrors(hipfftExecR2C(fftPlan, (hipfftReal*)_8bit_signal, tmp_result));
        
#ifndef JIT
        // Step 3
        ConvolveAndStoreTransposedC_Optimized<<<grid, block>>>(tmp_result, result, filter);
#else
        checkCudaErrors(cuFuncSetBlockShape(hKernel, TILE_DIM, BLOCK_ROWS, 1));
        int paramOffset = 0;
        SET_KERNEL_PARAM(hKernel, tmp_result, paramOffset);
        SET_KERNEL_PARAM(hKernel, result,     paramOffset);
        SET_KERNEL_PARAM(hKernel, filter,     paramOffset);
        
        // Launch the kernel (Driver API_)
        checkCudaErrors(cuLaunchGrid(hKernel, (COMPLEX_SIGNAL_SIZE + block.x - 1)/block.x, (BATCH_SIZE + block.y - 1)/block.y));
        printf("JIT CUDA kernel launched\n");
#endif
        checkCudaErrors(hipGetLastError());

    }

#ifndef JIT
    checkCudaErrors(hipEventRecord(end, 0));
    checkCudaErrors(hipEventSynchronize(end));
    checkCudaErrors(hipEventElapsedTime(&elapsedTime, start, end));
    printf("Time for the FFT: %fms\n", elapsedTime);
#endif
    checkCudaErrors(hipGetLastError());

    //Verify correct result    
    if(postprocess(reference, result, COMPLEX_SIGNAL_SIZE * BATCH_SIZE)) {
        printf("Verification successful.\n");
    } else {
        printf("!!! Verification Failed !!!\n");
    }

    //Cleanup
    checkCudaErrors(hipfftDestroy(fftPlan));

    checkCudaErrors(hipFree(_8bit_signal));
    checkCudaErrors(hipFree(result));
    checkCudaErrors(hipFree(filter));
    checkCudaErrors(hipFree(reference));
      
#ifdef JIT
    if (hModule)
    {
        checkCudaErrors(hipModuleUnload(hModule));
        hModule = 0;
    }
#endif
    
    //clean up driver state
    hipDeviceReset();

    printf("Done\n");
    
    return 0;
}