#include "hip/hip_runtime.h"
/* *********************************************************************** *
 *   WARSAW UNIVERSITY OF TECHNOLOGY                                       *
 *   FACULTY OF PHYSICS                                                    *
 *   NUCLEAR THEORY GROUP                                                  *
 *                                                                         *
 *   Author: Konrad Kobuszewski                                            *
 *                                                                         *
 *   This program is free software; you can redistribute it and/or modify  *
 *   it under the terms of the GNU General Public License as published by  *
 *   the Free Software Foundation; either version 2 of the License, or     *
 *   (at your option) any later version.                                   *
 *                                                                         *
 *   This program is distributed in the hope that it will be useful,       *
 *   but WITHOUT ANY WARRANTY; without even the implied warranty of        *
 *   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the         *
 *   GNU General Public License for more details.                          *
 *                                                                         *
 *   You should have received a copy of the GNU General Public License     *
 *   along with this program; if not, write to the                         *
 *   Free Software Foundation, Inc.,                                       *
 *   59 Temple Place - Suite 330, Boston, MA  02111-1307, USA.             *
 *                                                                         *
 * *********************************************************************** */ 
 

#include "Coulomb.hpp"

#include "kernels.cuh"



Coulomb::Coulomb(int devId, int nthreads):
                 _devId(devId)
{    
    //
    hipSetDevice(_devId);
    hipGetDeviceProperties(&dev_prop, _devId);
    dim3 _threads(nthreads);
    dim3 _blocks((int) ceilf( ((float) (NXYZ)) / nthreads ));
    gpu_threads = _threads;
    gpu_blocks  = _blocks;
    
    
    lcutoff = ((double) (NX+2*CX)) / (1.0 + sqrt(3.0)); // size of cube over 1 + 3
    cuErrCheck(  hipMemcpyToSymbol(HIP_SYMBOL(d_lcutoff), &lcutoff, sizeof(double))  );
    
    // allocate memory
    cuErrCheck(  hipMalloc((void**) &d_density,   (size_t) (NCX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(double))             );
    cuErrCheck(  hipMalloc((void**) &d_density_k, (size_t) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1) * sizeof(hipDoubleComplex))  );
    
    // create plans
    size_t wrk_sizes[4] = {0};
    cufftErrChk(  hipfftCreate(&plan_forward)              );
    cufftErrChk(  hipfftSetAutoAllocation(plan_forward,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_forward, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_D2Z, &wrk_sizes[0])  );
    
    cufftErrChk(  hipfftCreate(&plan_forward_cb)              );
    cufftErrChk(  hipfftSetAutoAllocation(plan_forward_cb,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_forward, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_D2Z, &wrk_sizes[1])  );
    
    cufftErrChk(  hipfftCreate(&plan_inverse)              );
    cufftErrChk(  hipfftSetAutoAllocation(plan_inverse,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_inverse, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_Z2D, &wrk_sizes[2])  );
    
    cufftErrChk(  hipfftCreate(&plan_inverse_cb)              );
    cufftErrChk(  hipfftSetAutoAllocation(plan_inverse_cb,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_inverse_cb, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_Z2D, &wrk_sizes[3])  );
    
    // allocate additional memory (work space for cufft)
    cuErrCheck(  hipMalloc((void**) &d_wrk_arr,  (size_t)  *std::max_element(wrk_sizes, &wrk_sizes[3]))  );
    cufftErrChk(  hipfftSetWorkArea(plan_forward,    d_wrk_arr)  );
    cufftErrChk(  hipfftSetWorkArea(plan_forward_cb, d_wrk_arr)  );
    cufftErrChk(  hipfftSetWorkArea(plan_inverse,    d_wrk_arr)  );
    cufftErrChk(  hipfftSetWorkArea(plan_inverse_cb, d_wrk_arr)  );
    
    // initialize constant memory
    double kkx[NCX];
    double kky[NCY];
    double kkz[NCZ];
    uint16_t i;  int j;
    for (i = 0; i <= NCX/2 - 1; i++) {  kkx[i] = 2. * M_PI/((double) NCX) * (double) i;     }  j = - i;
    for (i = NCX/2; i < NCX; i++)     {  kkx[i] = 2. * M_PI/((double) NCX) * (double) j; j++;}
    cuErrCheck(  hipMemcpyToSymbol(HIP_SYMBOL(d_kkx), kkx, NCX*sizeof(double))  );

    for (i = 0; i <= NCY/2 - 1; i++) {  kky[i] = 2. * M_PI/((double) NCY) * (double) i;     }  j = - i;
    for (i = NCY/2 ; i < NCY ; i++)   {  kky[i] = 2. * M_PI/((double) NCY) * (double) j; j++;}
    cuErrCheck(  hipMemcpyToSymbol(HIP_SYMBOL(d_kky), kky, NCY*sizeof(double))  );

    for (i = 0; i <= NCZ/2 - 1; i++) {  kkz[i] = 2. * M_PI/((double) NCZ) * (double) i;     }  j = - i;
    for (i = NCZ / 2; i < NCZ; i++)   {  kkz[i] = 2. * M_PI/((double) NCZ) * (double) j; j++;}
    cuErrCheck(  hipMemcpyToSymbol(HIP_SYMBOL(d_kkz), kkz, NCZ*sizeof(double))  );
    
    
    
#ifdef VERBOSE
    printf("Instance of class Coulomb.\n");
    printf("Compiled for lattice:  %ux%ux%u\n",NX,NY,NZ);
    printf("Computational lattice: %ux%ux%u\n",NX+2*CX,NY+2*CY,NZ+2*CZ);
    printf("Computational lattice: %ux%ux%u\n",NCX,NCY,NCZ);
    printf("Cutoff length lc:      %.3lf\n",lcutoff);
    
    for (unsigned ii=0; ii<4; ii++)
        printf("Plan %u: %lu\n",ii,wrk_sizes[ii]);
    
    printf("Mem allocated:\n");
    printf("    density (real space)    %lu\tB\n",(NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(double));
    printf("    density (recip. space)  %lu\tB\n",(NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex));
    printf("    total                   %lu\tB\n",(NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * (sizeof(hipDoubleComplex)+sizeof(double)) 
                                                                             + *std::max_element(wrk_sizes, &wrk_sizes[3]));
    
    printf("\n");
    printf("threads: (%u,%u,%u)\n",gpu_threads.x,gpu_threads.y,gpu_threads.z);
    printf("blocks:  (%u,%u,%u)\n",gpu_blocks.x,gpu_blocks.y,gpu_blocks.z);
    printf("\n");
    printf("\n");
#endif
}

Coulomb::~Coulomb()
{
    if (d_density != NULL) cuErrCheck(  hipFree(d_density)    );
    if (d_density != NULL) cuErrCheck(  hipFree(d_density_k)  );
    if (d_density != NULL) cuErrCheck(  hipFree(d_wrk_arr)    );
    
    //k_lattice_free();
    
    hipfftDestroy(plan_forward);
    hipfftDestroy(plan_forward_cb);
    hipfftDestroy(plan_inverse);
    hipfftDestroy(plan_inverse_cb);
}

void Coulomb::set_charge(const double charge)
{
    cuErrCheck(  hipMemcpyToSymbol(HIP_SYMBOL(d_charge), &charge, sizeof(double))  );
}


void Coulomb::get_density_enlarged(hipDoubleComplex* d_psi, double* h_result, double* h_t, const int _nthreads)
{
    dim3 _threads(_nthreads);
    dim3 _blocks((int) ceilf( ((float) (NX)*(NY)*(NZ)) / _nthreads ));
    if (_nthreads < 32) { _threads = gpu_threads; _blocks=gpu_blocks; }
    
    float rep;
    TIMER_CREATE(rep);
    TIMER_START(rep);
    
    // fill density array with zeros - TODO: check if it could be done once or move to kernel
    cuErrCheck(  hipMemset((void*)   d_density,   0,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(double))             );
    //cuErrCheck(  hipMemset((void*)   d_density_k, 0,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1) * sizeof(hipDoubleComplex))  );
    
    __kernel_enlarge_1Dindexing__<NX,NY,NZ,CX,CY,CZ><<<_blocks,_threads>>>(d_psi,d_density); // no errors in cuda-memcheck
    cuErrCheck( hipGetLastError() );
    TIMER_END(rep);
    
    if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(double), hipMemcpyDeviceToHost)  );
    if (h_t)         *h_t = (double) rep;
}


void Coulomb::get_vcoulomb_enlarged(hipDoubleComplex* h_result, double* h_t, int _nthreads, unsigned kernel_type)
{
    if (_nthreads < 32) _nthreads = 1024;
    
    
    float rep;
    TIMER_CREATE(rep);
    TIMER_START(rep);
    
    //if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    
    // fft forward
    cufftErrChk(  hipfftExecD2Z(plan_forward, (double*) d_density, (hipDoubleComplex*) d_density_k)  ); // HIPFFT_FORWARD
    
    // Coulomb kernel
    switch (kernel_type)
    {
        case 0:
        {
            dim3 _threads0(_nthreads);
            dim3 _blocks0((int) ceilf( ((float) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1)) / _nthreads ));
            kernel_coulomb_sph_cutoff0<NX+2*CX,NY+2*CY,NZ+2*CZ><<<_blocks0,_threads0>>>(d_density_k,d_density_k); break;
        }
        case 1:
        {
            dim3 _threads1(_nthreads);
            dim3 _blocks1((int) ceilf( ((float) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1)) / _nthreads ));
            kernel_coulomb_sph_cutoff1<NX+2*CX,NY+2*CY,NZ+2*CZ><<<_blocks1,_threads1>>>(d_density_k,d_density_k); break;
        }
        case 2:
        {
            dim3 _threads2(_nthreads);
            dim3 _blocks2((int) ceilf( ((float) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1)) / _nthreads ));
            kernel_coulomb_sph_cutoff2<NX+2*CX,NY+2*CY,NZ+2*CZ><<<_blocks2,_threads2>>>(d_density_k,d_density_k); break;
        }
        case 3:
        {
            dim3 _threads3(_nthreads);
            dim3 _blocks3((int) ceilf( ((float) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1)) / _nthreads ));
            kernel_coulomb_sph_cutoff_cnst0<NX+2*CX,NY+2*CY,NZ+2*CZ><<<_blocks3,_threads3>>>(d_density_k,d_density_k); break;
        }
        case 4:
        {
            dim3 _threads3(_nthreads);
            dim3 _blocks3((int) ceilf( ((float) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1)/2) / _nthreads ));
            kernel_coulomb_sph_cutoff_cnst1<NX+2*CX,NY+2*CY,NZ+2*CZ><<<_blocks3,_threads3>>>(d_density_k,d_density_k); break;
        }
        case 5:
        {
            dim3 _threads3(_nthreads);
            dim3 _blocks3((int) ceilf( ((float) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1)/4) / _nthreads ));
            kernel_coulomb_sph_cutoff_cnst2<NX+2*CX,NY+2*CY,NZ+2*CZ><<<_blocks3,_threads3>>>(d_density_k,d_density_k); break;
        }
    }
    
    
    //if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density_k,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    
    // fft inverse
    cufftErrChk(  hipfftExecZ2D(plan_inverse, (hipDoubleComplex*) d_density_k, (double*) d_density)  ); // HIPFFT_BACKWARD
    TIMER_END(rep);
    
    if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density_k,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    if (h_t)         *h_t = (double) rep;
}


void Coulomb::get_vcoulomb_lessened(double* d_vcoulomb, double* h_result, double* h_t, const int _nthreads)
{
    dim3 _threads(_nthreads);
    dim3 _blocks((int) ceilf( ((float) (NX)*(NY)*(NZ)) / _nthreads ));
    if (_nthreads < 32) { _threads = gpu_threads; _blocks=gpu_blocks; }
    
    float rep;
    TIMER_CREATE(rep);
    TIMER_START(rep);
    
    __kernel_lessen_1Dindexing__<NX,NY,NZ,CX,CY,CZ><<<_blocks,_threads>>>(d_density,d_vcoulomb); // no errors in cuda-memcheck
    cuErrCheck( hipGetLastError() );
    TIMER_END(rep);
    
    if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_vcoulomb,(size_t) NX*NY*NZ * sizeof(double), hipMemcpyDeviceToHost)  );
    if (h_t)         *h_t = (double) rep;
}

void Coulomb::save_info(const double sigma)
{
    // save info
    FILE* file_info = fopen("coulomb.info","w");
    fprintf(file_info,"nx\t%u\n",NX);
    fprintf(file_info,"ny\t%u\n",NY);
    fprintf(file_info,"nz\t%u\n",NZ);
    fprintf(file_info,"cx\t%u\n",CX);
    fprintf(file_info,"cy\t%u\n",CY);
    fprintf(file_info,"cz\t%u\n",CZ);
    fprintf(file_info,"aho\t%lf\n",sigma);
    fclose(file_info);
}

void Coulomb::get_lattice(unsigned* nx, unsigned* ny, unsigned* nz, unsigned* cx, unsigned* cy, unsigned* cz, double* _lcutoff)
{
    if (nx)       *nx = NX;
    if (ny)       *ny = NY;
    if (nz)       *nz = NZ;
    if (cx)       *cx = CX;
    if (cy)       *cy = CY;
    if (cz)       *cz = CZ;
    if (_lcutoff) *_lcutoff = lcutoff;
}

void Coulomb::get_lattice(unsigned* lattice, double* _lcutoff)
{
    this->get_lattice(lattice+0,lattice+1,lattice+2,lattice+3,lattice+4,lattice+5,_lcutoff);
}


































//  ======================================== 
/*
void Coulomb::get_vcoulomb_enlarged_idx3d(hipDoubleComplex* h_result, double* h_t, int _nthreads)
{
    int nx_threads = 1;
    int ny_threads = 2;
#if ((NZ+2*CZ) > 32)
    int nz_threads = 64;
#else
    int nz_threads = 32;
#endif
    dim3 gpu_threads(nx_threads,ny_threads,nz_threads);
    dim3 gpu_blocks( (NX + 2*CX + nx_threads -1)/nx_threads,
                     (NY + 2*CY + ny_threads -1)/ny_threads,
                     (NZ/2 + CZ + 1 + nz_threads -1)/nz_threads ); // NOTE: run NZ/2 threads in z direction and not NZ/2+1
    //printf("threads: %dx%dx%d\t\tblocks: %dx%dx%d\t\ttotal: \n",_threads.x,_threads.y,_threads.z,_blocks.x,_blocks.y,_blocks.z);
    
    float rep;
    TIMER_CREATE(rep);
    TIMER_START(rep);
    
    //if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    
    // fft forward
    cufftErrChk(  hipfftExecD2Z(plan_forward, (double*) d_density, (hipDoubleComplex*) d_density_k)  ); // HIPFFT_FORWARD
    
    // Coulomb kernel
    //kernel_coulomb_sph_cutoff_3Didx<NX+2*CX,NY+2*CY,NZ+2*CZ><<<gpu_blocks,gpu_threads>>>(d_density_k,d_density_k); // NOTE: run NZ/2 threads in z direction and not NZ/2+1
    kernel_coulomb_sph_cutoff_3Didx0<NX+2*CX,NY+2*CY,NZ+2*CZ><<<gpu_blocks,gpu_threads>>>(d_density_k,d_density_k);
    cuErrCheck( hipGetLastError() );
    
    // fft inverse
    cufftErrChk(  hipfftExecZ2D(plan_inverse, (hipDoubleComplex*) d_density_k, (double*) d_density)  ); // HIPFFT_BACKWARD
    TIMER_END(rep);
    
    if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density_k,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ/2+CZ+1) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    if (h_t)         *h_t = (double) rep;
}
*/