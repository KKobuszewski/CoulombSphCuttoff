#include "hip/hip_runtime.h"
#include "Coulomb.hpp"

#include "kernels.cuh"



Coulomb::Coulomb(int devId, int nthreads):
                 _devId(devId)
{    
    //
    hipSetDevice(_devId);
    hipGetDeviceProperties(&dev_prop, _devId);
    dim3 _threads(nthreads);
    dim3 _blocks((int) ceilf( ((float) (NXYZ)) / nthreads ));
    gpu_threads = _threads;
    gpu_blocks  = _blocks;
    
    
    lcutoff = ((double) (NX+2*CX)) / (1.0 + sqrt(3.0)); // size of cube over 1 + 3
    cuErrCheck(  hipMemcpyToSymbol(HIP_SYMBOL(d_lcutoff), &lcutoff, sizeof(double))  );
    
    // allocate memory
    cuErrCheck(  hipMalloc((void**) &d_density,   (size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex))  );
    cuErrCheck(  hipMalloc((void**) &d_density_k, (size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex))  );
    
    // fill density array with zeros - TODO: check if it could be done once or move to kernel
    cuErrCheck(  hipMemset((void*)   d_density,   0,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex))  );
    cuErrCheck(  hipMemset((void*)   d_density_k, 0,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex))  );
    
    // create plans
    size_t wrk_sizes[4] = {0};
    cufftErrChk(  hipfftCreate(&plan_forward)               );
    cufftErrChk(  hipfftSetAutoAllocation(plan_forward,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_forward, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_Z2Z, &wrk_sizes[0])  );
    
    cufftErrChk(  hipfftCreate(&plan_forward_cb)               );
    cufftErrChk(  hipfftSetAutoAllocation(plan_forward_cb,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_forward, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_Z2Z, &wrk_sizes[1])  );
    
    cufftErrChk(  hipfftCreate(&plan_inverse)               );
    cufftErrChk(  hipfftSetAutoAllocation(plan_inverse,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_inverse, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_Z2Z, &wrk_sizes[2])  );
    
    cufftErrChk(  hipfftCreate(&plan_inverse_cb)               );
    cufftErrChk(  hipfftSetAutoAllocation(plan_inverse_cb,0)  );
    cufftErrChk(  hipfftMakePlan3d(plan_inverse_cb, NX+2*CX, NY+2*CY, NZ+2*CZ, HIPFFT_Z2Z, &wrk_sizes[3])  );
    
    // allocate additional memory (work space for cufft)
    cuErrCheck(  hipMalloc((void**) &d_wrk_arr,  (size_t)  *std::max_element(wrk_sizes, &wrk_sizes[3]))  );
    cufftErrChk(  hipfftSetWorkArea(plan_forward,    d_wrk_arr)  );
    cufftErrChk(  hipfftSetWorkArea(plan_forward_cb, d_wrk_arr)  );
    cufftErrChk(  hipfftSetWorkArea(plan_inverse,    d_wrk_arr)  );
    cufftErrChk(  hipfftSetWorkArea(plan_inverse_cb, d_wrk_arr)  );
    
    //k_lattice_init();
    
#ifdef VERBOSE
    printf("Instance of class Coulomb.\n");
    printf("Compiled for lattice:  %ux%ux%u\n",NX,NY,NZ);
    printf("Computational lattice: %ux%ux%u\n",NX+2*CX,NY+2*CY,NZ+2*CZ);
    printf("Cutoff length lc:      %.3lf\n",lcutoff);
    
    for (unsigned ii=0; ii<4; ii++)
        printf("Plan %u: %lu\n",ii,wrk_sizes[ii]);
    
    printf("Mem allocated:\n");
    printf("    density (real space)    %lu\tB\n",(NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex));
    printf("    density (recip. space)  %lu\tB\n",(NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex));
    printf("    total                   %lu\tB\n",(NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * (sizeof(hipDoubleComplex)*2) 
                                                                + *std::max_element(wrk_sizes, &wrk_sizes[3]));
    
    printf("\n");
    printf("threads: (%u,%u,%u)\n",gpu_threads.x,gpu_threads.y,gpu_threads.z);
    printf("blocks:  (%u,%u,%u)\n",gpu_blocks.x,gpu_blocks.y,gpu_blocks.z);
    printf("\n");
    printf("\n");
#endif
}

Coulomb::~Coulomb()
{
    if (d_density != NULL) cuErrCheck(  hipFree(d_density)    );
    if (d_density != NULL) cuErrCheck(  hipFree(d_density_k)  );
    if (d_density != NULL) cuErrCheck(  hipFree(d_wrk_arr)    );
    
    //k_lattice_free();
    
    hipfftDestroy(plan_forward);
    hipfftDestroy(plan_forward_cb);
    hipfftDestroy(plan_inverse);
    hipfftDestroy(plan_inverse_cb);
}

void Coulomb::set_charge(const double charge)
{
    cuErrCheck(  hipMemcpyToSymbol(HIP_SYMBOL(d_charge), &charge, sizeof(double))  );
}


void Coulomb::get_density_enlarged(hipDoubleComplex* d_psi, hipDoubleComplex* h_result, double* h_t)
{
    float rep;
    TIMER_CREATE(rep);
    TIMER_START(rep);
    __kernel_enlarge_1Dindexing__<NX,NY,NZ,CX,CY,CZ><<<gpu_blocks,gpu_threads>>>(d_psi,d_density); // no errors in cuda-memcheck
    cuErrCheck( hipGetLastError() );
    TIMER_END(rep);
    
    if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    if (h_t)         *h_t = (double) rep;
}


void Coulomb::get_vcoulomb_enlarged(hipDoubleComplex* h_result, double* h_t)
{
    unsigned nthreads = 1024;
    dim3 _threads(nthreads);
    dim3 _blocks((int) ceilf( ((float) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ)) / nthreads ));
    
    printf("\n");
    printf("threads: (%u,%u,%u)\n",_threads.x,_threads.y,_threads.z);
    printf("blocks:  (%u,%u,%u)\n",_blocks.x,_blocks.y,_blocks.z);
    printf("\n");
    
    float rep;
    TIMER_CREATE(rep);
    TIMER_START(rep);
    
    // fft forward
    cufftErrChk(  hipfftExecZ2Z(plan_forward, (hipDoubleComplex*) d_density, (hipDoubleComplex*) d_density_k, HIPFFT_FORWARD)  ); // HIPFFT_FORWARD
    
    //if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density_k,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    
    // Coulomb kernel
    // truncating last dimension of array due to real to complex FFT symmetry
    kernel_coulomb_sph_cutoff<NX+2*CX,NY+2*CY,NZ+2*CZ><<<_blocks,_threads>>>(d_density_k,d_density_k);
    
    
    if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density_k,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    
    // fft inverse
    cufftErrChk(  hipfftExecZ2Z(plan_inverse, (hipDoubleComplex*) d_density_k, d_density, HIPFFT_BACKWARD)  ); // HIPFFT_BACKWARD
    TIMER_END(rep);
    
    //if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_density_k,(size_t) (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    if (h_t)         *h_t = (double) rep;
}

void Coulomb::get_vcoulomb_lessened(double* d_vcoulomb, double* h_result, double* h_t)
{
    float rep;
    TIMER_CREATE(rep);
    TIMER_START(rep);
    
    __kernel_lessen_1Dindexing__<NX,NY,NZ,CX,CY,CZ><<<gpu_blocks,gpu_threads>>>(d_density,d_vcoulomb); // no errors in cuda-memcheck
    cuErrCheck( hipGetLastError() );
    TIMER_END(rep);
    
    if (h_result)     cuErrCheck(  hipMemcpy(h_result, d_vcoulomb,(size_t) NX*NY*NZ * sizeof(double), hipMemcpyDeviceToHost)  );
    if (h_t)         *h_t = (double) rep;
}


void Coulomb::save_info(const double sigma)
{
    // save info
    FILE* file_info = fopen("data/coulomb.info","w");
    fprintf(file_info,"nx\t%u\n",NX);
    fprintf(file_info,"ny\t%u\n",NY);
    fprintf(file_info,"nz\t%u\n",NZ);
    fprintf(file_info,"cx\t%u\n",CX);
    fprintf(file_info,"cy\t%u\n",CY);
    fprintf(file_info,"cz\t%u\n",CZ);
    fprintf(file_info,"aho\t%lf\n",sigma);
    fclose(file_info);
}