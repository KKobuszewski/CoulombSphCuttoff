#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>
#include <thrust/functional.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>


#include <cuerror.cuh>
#include <reductions.cuh>
#include <thread_fence_reductions.cuh>
#include <cub_utils.cuh>


#ifndef NX
#define NX 128
#endif

#ifndef NY
#define NY 128
#endif

#ifndef NZ
#define NZ 128
#endif

#define NXYZ (NX*NY*NZ)

inline double gaussian3D(double x, double y,double z, double sigma)
{
    return exp( -1.*(x*x + y*y + z*z)/( 2.*sigma*sigma) );
}

inline void fill_array(double *psi, double *params)
{
    #pragma omp parallel for num_threads(8) 
    for (int ix = 0; ix < NX; ix++)  
    for (int iy = 0; iy < NY; iy++)  
    for (int iz = 0; iz < NZ; iz++) 
    {
        psi[iz + NZ*(iy + NY*ix)] = gaussian3D((double) ix-NX/2,(double) iy-NY/2,(double) iz-NZ/2,params[0]);
    }
}



template<typename T>
inline void save_file(const char* filename, void* data, size_t size)
{
    FILE* file = fopen(filename,"wb");
    fwrite(data,(size_t) size * sizeof(T),1,file);
    fclose(file);
}

int main(int argc, char* argv[])
{
    double *h_psi; // used as a buffer on host side, be careful of 'magic' with pointer casting
    double *d_psi, *d_wrk_mem;
    cuErrCheck(  hipHostMalloc((void**) &h_psi,(size_t)  NXYZ * sizeof(double))  );
    cuErrCheck(  hipMalloc((void**) &d_psi,(size_t)  NXYZ * sizeof(double))  );
    cuErrCheck(  hipMalloc((void**) &d_wrk_mem,(size_t)  NXYZ * sizeof(double))  );
    
    
    
    
    // get analytical result
    double sigma = 5.0; // a_ho, charge
    fill_array(h_psi,&sigma);
    //save_file<double>("orginal.bin",(void*) h_psi, NX*NY*NZ );
    cuErrCheck(  hipMemcpy(d_psi, h_psi, (size_t) NXYZ * sizeof(double), hipMemcpyHostToDevice)  );
    
    
    double result = 0.0;
    
    printf("Threadfence reduce\n");
    for (int it=0; it<100; it++)
    result = threadfence_reduce<double>(d_psi, d_wrk_mem, NXYZ);
    printf("result: %lf   /    %lf\n",result,pow( 2.*sigma*sigma*M_PI, 1.5 ) );
    printf("\n");
    
    printf("Simple reduce\n");
    for (int it=0; it<100; it++)
    result = block_reduce<double>(d_psi, NXYZ, d_wrk_mem);
    printf("result: %lf   /    %lf\n",result,pow( 2.*sigma*sigma*M_PI, 1.5 ) );
    printf("\n");
    
    printf("CUB reduce\n");
    for (int it=0; it<100; it++)
    //result = cub_reduce<double>(d_psi, NXYZ, d_wrk_mem);
    result = cub_reduce<double>(d_psi, NXYZ);
    printf("result: %lf   /    %lf\n",result,pow( 2.*sigma*sigma*M_PI, 1.5 ) );
    printf("\n");
    
    printf("Thrust reduce\n");
    for (int it=0; it<100; it++)
    result = thrust::reduce(thrust::device_pointer_cast<double>(d_psi),
                            thrust::device_pointer_cast<double>(d_psi+NXYZ),
                            (double) 0.0, thrust::plus<double>());
    printf("result: %lf   /    %lf\n",result,pow( 2.*sigma*sigma*M_PI, 1.5 ) );
    printf("\n");
    
    
    hipFree(d_psi);
    hipHostFree(h_psi);
    
    return EXIT_SUCCESS;
}