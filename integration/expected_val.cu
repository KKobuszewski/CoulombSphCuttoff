#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>
//#include <thrust/functional.h>
//#include <thrust/transform_reduce.h>

#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <complex.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>



#include <cuerror.cuh>
#include <cutimer.h>
//#include <reductions.cuh>
#include <cub_utils.cuh>
#include <Integration.hpp>


#ifndef NX
#define NX 128
#endif

#ifndef NY
#define NY 128
#endif

#ifndef NZ
#define NZ 128
#endif

#define NXYZ (NX*NY*NZ)

/*
 * Calculation of integral 
 * Integrate[
 *   Exp[-((x^2 + y^2 + z^2)/\[Sigma]^2)] (x /\[Sigma])^2, 
 *   {x, -Infinity, Infinity}, {y, -Infinity, Infinity}, {z, -Infinity, Infinity}
 * ]
 * The result should be 0.5 pi^3/2 sigma^3
 */

inline __complex__ double gaussian3D(double x, double y,double z, double sigma)
{
    return cexp( -1.*(x*x + y*y + z*z)/( 2.*sigma*sigma) );
}

inline void fill_array(__complex__ double *psi, double *params)
{
    printf("%lf\n",params[0]);
    #pragma omp parallel for num_threads(8) 
    for (int ix = 0; ix < NX; ix++)  
    for (int iy = 0; iy < NY; iy++)  
    for (int iz = 0; iz < NZ; iz++)
    {
        //psi[iz + NZ*(iy + NY*ix)] = 1.0 + I*0.;
        psi[iz + NZ*(iy + NY*ix)] = gaussian3D((double) ix-NX/2,(double) iy-NY/2,(double) iz-NZ/2,params[0]); //  
        //if (iz == 1) printf("%lf+%lfj\n",creal(psi[iz + NZ*(iy + NY*ix)]), cimag(psi[iz + NZ*(iy + NY*ix)]) );
    }
}

inline void potential(double *V, double *params)
{
    #pragma omp parallel for num_threads(8) 
    for (int ix = 0; ix < NX; ix++)
    for (int iy = 0; iy < NY; iy++)
    for (int iz = 0; iz < NZ; iz++)
    {
        //V[iz + NZ*(iy + NY*ix)] = ((double) (ix-NX/2 + iy-NY/2 + iz-NZ/2))/params[0]; // 0
        V[iz + NZ*(iy + NY*ix)] = 1.0; // times to
        //V[iz + NZ*(iy + NY*ix)] = ( ((double)ix-NX/2) * ((double)ix-NX/2) )/params[0]/params[0]; // what printed
    }
}

template<typename T>
inline void save_file(const char* filename, void* data, size_t size)
{
    FILE* file = fopen(filename,"wb");
    fwrite(data,(size_t) size * sizeof(T),1,file);
    fclose(file);
}

int main(int argc, char* argv[])
{
    __complex__ double *h_psi; // used as a buffer on host side, be careful of 'magic' with pointer casting
    hipDoubleComplex *d_psi;
    double* h_entity, *d_entity;
    cuErrCheck(  hipHostMalloc((void**) &h_psi,(size_t)  NXYZ * sizeof(__complex__  double))  );
    cuErrCheck(  hipHostMalloc((void**) &h_entity,(size_t)  NXYZ * sizeof(double))    );
    cuErrCheck(  hipMalloc((void**) &d_psi,(size_t)  NXYZ * sizeof(hipDoubleComplex))  );
    cuErrCheck(  hipMalloc((void**) &d_entity,(size_t)  NXYZ * sizeof(double))        );
    
    // initialize Integration module
    Integration<NX,NY,NZ>* integ = new Integration<NX,NY,NZ>();
    
    
    // get analytical result
    double sigma = 5.0; // a_ho, charge
    fill_array(h_psi,&sigma);
    potential(h_entity,&sigma);
    save_file<__complex__ double>("orginal.bin",(void*) h_psi, NX*NY*NZ );
    cuErrCheck(  hipMemcpy(d_psi, h_psi, (size_t) NX*NY*NZ * sizeof(hipDoubleComplex), hipMemcpyHostToDevice)  );
    cuErrCheck(  hipMemcpy(d_entity, h_entity, (size_t) NX*NY*NZ * sizeof(double), hipMemcpyHostToDevice)  );
    
    
    double val = 0.0;
    
    printf("Integrating custom...\n");
    for (int it=0; it<100; it++)
        val = integ->get_exp_val_rect_custom(d_entity, d_psi);
    printf( "%e\t/\t%e\n", val, 0.5*pow(sqrt(M_PI)*sigma,3) );
    printf("\n");
    
    
    printf("Integrating custom...\n");
    for (int it=0; it<100; it++)
        val = integ->get_exp_val_rect_threadfence(d_entity,d_psi,128);
    printf( "%e\t/\t%e\n", val, 0.5*pow(sqrt(M_PI)*sigma,3) );
    printf("\n");
    
    
    printf("Integrating Thrust...\n");
    for (int it=0; it<100; it++)
        val = integ->get_exp_val_rect_thrust(d_entity,d_psi);
    printf( "%e\t/\t%e\n", val, 0.5*pow(sqrt(M_PI)*sigma,3) );
    printf("\n");
    
    
    
    printf("Integrating CUBLAS...\n");
    hipDoubleComplex c_val;
    for (int it=0; it<100; it++)
        c_val = integ->get_exp_val_rect_cublas(d_entity,d_psi);
    printf( "%lf+%lfj\t/\t%lf\n", c_val.x, c_val.y, 0.5*pow(sqrt(M_PI)*sigma,3) );
    printf("\n");
    
    delete integ;
    hipFree(d_psi);
    hipFree(d_entity);
    hipHostFree(h_psi);
    hipHostFree(h_entity);
    
    return EXIT_SUCCESS;
}