#include "hip/hip_runtime.h"
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/iterator/counting_iterator.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h>

#include <Integration.hpp>


#ifndef NX
#define NX 128
#endif

#ifndef NY
#define NY 128
#endif

#ifndef NZ
#define NZ 128
#endif

#define NXYZ (NX*NY*NZ)

inline __complex__ double gaussian3D(double x, double y,double z, double sigma)
{
    return cexp( -1.*(x*x + y*y + z*z)/( 2.*sigma*sigma) );
    //return 1. + I*0.;
}

inline void fill_array(__complex__ double *psi, double *params)
{
    #pragma omp parallel for num_threads(8) 
    for (int ix = 0; ix < NX; ix++)  
    for (int iy = 0; iy < NY; iy++)  
    for (int iz = 0; iz < NZ; iz++) 
    {
        psi[iz + NZ*(iy + NY*ix)] = gaussian3D((double) ix-NX/2,(double) iy-NY/2,(double) iz-NZ/2,params[0]) + I*0.;
    }
}





template<typename T>
inline void save_file(const char* filename, void* data, size_t size)
{
    FILE* file = fopen(filename,"wb");
    fwrite(data,(size_t) size * sizeof(T),1,file);
    fclose(file);
}

int main(int argc, char* argv[])
{
    __complex__ double *h_psi; // used as a buffer on host side, be careful of 'magic' with pointer casting
    hipDoubleComplex *d_psi;
    cuErrCheck(  hipHostMalloc((void**) &h_psi,(size_t)  NXYZ * sizeof(__complex__  double))  );
    cuErrCheck(  hipMalloc((void**) &d_psi,(size_t)  NXYZ * sizeof(hipDoubleComplex))  );
    
    
    
    // initialize Integration module
    Integration<NX,NY,NZ>* integ = new Integration<NX,NY,NZ>();
    
    
    // get analytical result
    double sigma = 5.0; // a_ho, charge
    fill_array(h_psi,&sigma);
    save_file<__complex__ double>("orginal.bin",(void*) h_psi, NX*NY*NZ );
    cuErrCheck(  hipMemcpy(d_psi, h_psi, (size_t) NX*NY*NZ * sizeof(hipDoubleComplex), hipMemcpyHostToDevice)  );
    
    
    double norm = 0.0;
    norm = integ->normalize_cublas(d_psi);
    
    printf("psi(0,0): %lf\n",creal(h_psi[NZ/2 + NZ*(NY/2 + NY*NX/2)]));
    printf("norm^2= %lf\t\t(sigma*sigma*pi)^(3/2): %lf\n",norm*norm, pow( sigma*sigma*M_PI, 1.5 ) ); // norm of square of gaussian!
    // for sigma = 10  ->  15749.609945722419
    printf("%lf\n",integ->get_norm_cublas(d_psi));
    cuErrCheck(  hipMemcpy(h_psi, d_psi, (size_t) NX*NY*NZ * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost)  );
    
    
    printf("psi(0,0): %lf\t%lf\n",creal(h_psi[NZ/2 + NZ*(NY/2 + NY*NX/2)]), 1./norm);
    
    delete integ;
    hipFree(d_psi);
    hipHostFree(h_psi);
    
    return EXIT_SUCCESS;
}