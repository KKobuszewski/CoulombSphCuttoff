#include "hip/hip_runtime.h"
// needed for thrust compilation ...
#include <thrust/device_ptr.h>
#include <thrust/device_vector.h>
#include <thrust/inner_product.h>
#include <thrust/functional.h>
#include <thrust/extrema.h>

#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <complex.h>
#include <stdint.h>

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hip/hip_complex.h> 

// custom includes
#include <utils/compare_thrust.cuh>
#include <Coulomb/Coulomb.hpp>

#define SQRT2  ((double) sqrt(2.))
#define SQRTPI ((double) 1.7724538509055159)



inline __complex__ double gaussian3D(double x, double y,double z, double sigma)
{
    return cexp( -1.*(x*x + y*y + z*z)/( 2.*sigma*sigma) );
    //return 1. + I*0.;
}

inline void fill_array(__complex__ double *psi, double *params)
{
    #pragma omp parallel for num_threads(8) 
    for (int ixyz = 0; ixyz < NXYZ; ixyz++) 
    {  
        int ix,iy,iz,i;
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        psi[ixyz] = pow(1./SQRTPI/params[0],1.5) *
                    gaussian3D((double) ix-NX/2.,(double) iy-NY/2.,(double) iz-NZ/2.,params[0]) + I*0.;
    }
}

inline void analytical(double* potential, double* params)
{
    #pragma omp parallel for num_threads(8) 
    for (int ixyz = 0; ixyz < NXYZ; ixyz++) 
    {  
        int ix,iy,iz,i;
        ixyz2ixiyiz(ixyz,ix,iy,iz,i);
        const double r = sqrt( pow((double) ix-NX/2.,2) + pow((double) iy-NY/2.,2) + pow((double) iz-NZ/2.,2) );
        potential[ixyz] = (r == 0) ? params[1]*params[1] / ( 2.0 * SQRTPI*SQRTPI*SQRTPI * params[0]) : 
                                     params[1]*params[1] * erf( r / params[0] ) / (4. * M_PI * r);
    }
}

template<typename T>
inline void save_file(const char* filename, void* data, size_t size)
{
    FILE* file = fopen(filename,"wb");
    fwrite(data,(size_t) size * sizeof(T),1,file);
    fclose(file);
}

int main(int argc, char* argv[])
{
    __complex__ double *h_psi; // used as a buffer on host side, be careful of 'magic' with pointer casting
    double* h_analytical;
    hipDoubleComplex *d_psi;
    double *d_vcoulomb, *d_analytical;
    cuErrCheck(  hipHostMalloc((void**) &h_psi,(size_t)  (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) * sizeof(__complex__  double))  );
    cuErrCheck(  hipHostMalloc((void**) &h_analytical,(size_t)  NX*NY*NZ * sizeof(double))  );
    cuErrCheck(  hipMalloc((void**) &d_psi,(size_t)  NX*NY*NZ * sizeof(hipDoubleComplex))  );
    cuErrCheck(  hipMalloc((void**) &d_vcoulomb,(size_t)  NX*NY*NZ * sizeof(double))  );
    cuErrCheck(  hipMalloc((void**) &d_analytical,(size_t)  NX*NY*NZ * sizeof(double))  );
    
    
    printf("cx: %u %u %u\n",CX,CY,CZ);
    printf("Nx: %ux%ux%u\r%lu\n",NX,NY,NZ,NXYZ);
    
    // initialize Coulomb
    double sigma = 3.;    // rather aho
    double charge = 1.0;
    Coulomb* c = new Coulomb();
    c->set_charge(charge);
    double lcutoff = 0.;
    unsigned lattice[6] = {0};
    c->get_lattice(lattice, &lcutoff);
    
    
    // get analytical result
    double params[2] = { sigma, charge }; // a_ho, charge
    analytical(h_analytical,params);
    save_file<double>("analytical.bin",(void*) h_analytical, NX*NY*NZ );
    cuErrCheck(  hipMemcpy(d_analytical, h_analytical, (size_t) NX*NY*NZ * sizeof(double), hipMemcpyHostToDevice)  );
    
    
    // init timing file
    char timing_filename[256];
    snprintf(timing_filename, 256, "timing%ux%ux%u.dat", lattice[0], lattice[1], lattice[2]);
    FILE* file_timing = fopen(timing_filename,"w");
    fprintf(file_timing,"nx\tny\tnz\tthreads\tenlarging\t\t\tcoulomb\t\t\tlessening\n");
    
    unsigned threads = 1024;
    //for (unsigned threads = 32; threads <= 1024; threads += 32)
    //{
        printf("Threads: %u\n",threads);
        
        // timing
        double mean_time_enlarging = 0.0;
        double mean_time_coulomb   = 0.0;
        double mean_time_lessening = 0.0;
        double var_time_enlarging = 0.0;
        double var_time_coulomb   = 0.0;
        double var_time_lessening = 0.0;
        
        unsigned iters = 100;
        for (unsigned it = 0; it < iters; it++)
        {
            // fill orginal array with values
            fill_array(h_psi,&sigma);
            cuErrCheck(  hipMemcpy( d_psi, h_psi, (size_t) NX*NY*NZ * sizeof(hipDoubleComplex), hipMemcpyHostToDevice)  );
            if (it == 0) save_file<hipDoubleComplex>("orginal3D.bin",(void*) h_psi, NX*NY*NZ);
            
            // timing
            double time_enlarging = 0.0;
            double time_coulomb   = 0.0;
            double time_lessening = 0.0;
            
            // get density from given wavefunction and save to enlarged array
            c->get_density_enlarged(d_psi,(double*) h_psi,&time_enlarging,threads);
            //if (it == 0) save_file<double>("data/resized3D.bin",(void*) h_psi, (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ) );
            
            // perform FFTs and convole with Coulomb kernel
            //c->get_vcoulomb_enlarged((hipDoubleComplex*) h_psi, &time_coulomb,threads);
            c->get_vcoulomb_enlarged_idx3d((hipDoubleComplex*) h_psi, &time_coulomb,threads);
            //if (it == 0) save_file<hipDoubleComplex>("data/density_transform3D.bin",(void*) h_psi, (NX+2*CX)*(NY+2*CY)*(NZ+2*CZ));
            
            // truncate vcoulomb array to size of orginal array and save to given location
            c->get_vcoulomb_lessened(d_vcoulomb, (double*) h_psi, &time_lessening,threads);
            if (it == 0) save_file<double>("vcoulomb.bin",(void*) h_psi, NXYZ);
            
            int differences = 0;
            differences = thrust_compare_arrays<double,-7>(d_vcoulomb, d_analytical, (size_t) NX*NY*NZ);
            if (differences) printf("WARNING: Number of differences: %d\n",differences);
            
            mean_time_enlarging += time_enlarging;
            mean_time_coulomb   += time_coulomb;
            mean_time_lessening += time_lessening;
            var_time_enlarging  += time_enlarging*time_enlarging;
            var_time_coulomb    += time_coulomb*time_coulomb;
            var_time_lessening  += time_lessening*time_lessening;
        }
        
        mean_time_enlarging /= iters;
        mean_time_coulomb   /= iters;
        mean_time_lessening /= iters;
        var_time_enlarging = (var_time_enlarging - mean_time_enlarging*mean_time_enlarging)/(iters - 1);
        var_time_coulomb   = (var_time_coulomb   - mean_time_coulomb*mean_time_coulomb)/(iters - 1);
        var_time_lessening = (var_time_lessening - mean_time_lessening*mean_time_lessening)/(iters - 1);
        
        fprintf(file_timing,"%u\t%u\t%u\t%u\t%.6e\t%.6e\t%.6e\t%.6e\t%.6e\t%.6e\n",
                            lattice[0],lattice[1],lattice[2],threads,
                            mean_time_enlarging,var_time_enlarging,
                            mean_time_coulomb,var_time_coulomb,
                            mean_time_lessening,var_time_lessening);
        //printf("\n\n");
    //}
    
    c->save_info(sigma);
    
    // close files
    fclose(file_timing);
    
    // clean memory
    delete c;
    hipFree(d_psi);
    hipFree(d_vcoulomb);
    hipFree(d_analytical);
    hipHostFree(h_psi);
    hipHostFree(h_analytical);
    
    return EXIT_SUCCESS;
}